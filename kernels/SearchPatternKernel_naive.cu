
#include <hip/hip_runtime.h>


extern "C"
__global__
void SearchPatternKernel_naive(int *d_nFound, int *d_offsets, int nMaxMatched,
							   const unsigned char *d_pattern, int patternLength,
                               const unsigned char *d_text, int searchLength) {

    int gid = blockDim.x * blockIdx.x + threadIdx.x;
    if (gid < searchLength) {
        const unsigned char *d_myPos = &d_text[gid];
        int idx = 0;
        for (; idx < patternLength; ++idx) {
            if (d_pattern[idx] != d_myPos[idx])
                break;
        }

        if (idx == patternLength) {
            int offsetPos = atomicAdd(d_nFound, 1);
            if (offsetPos < nMaxMatched)
            	d_offsets[offsetPos] = gid;
        }
    }
}
